
#include <hip/hip_runtime.h>
__global__ void funkcija(float* m, int *V1, int *k1){
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    int V = V1[0];
    int k = k1[0];
    if (i < V && j < V){
        float t1 = m[i*V + k] + m[k*V + j];
        float t2 = m[i*V + j];

        m[i*V + j] = (t1 < t2) ? t1 : t2;
    }
}