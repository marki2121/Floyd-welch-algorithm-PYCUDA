#include "hip/hip_runtime.h"
__global__ void funkcija(float *rez, float* m, int *k){
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (m(i, k) + m(k, j) < m(i, j)){
        m(i, j) = m(i, k) + m(k, j);
    }

    rez = m;
}
