
#include <hip/hip_runtime.h>
/*__device__ void calc(float* m, int V, int k, int i, int j){
    if((i >= V) || (j >= V) || (k >=V)) return;
    
    const unsigned int kj = k*V + j; 
    const unsigned int ij = i*V + j;
    const unsigned int ki = i*V + k;

    float t1 = m[ki] + m[kj];
    float t2 = m[ij];

    m[ij] = (t1 < t2) ? t1 : t2;
}*/

__global__ void funkcija(float* m, int *V1, int *k1){
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    int V = V1[0];
    int k = k1[0];
    
    if (i < V && j < V){
        float t1 = m[i*V + k] + m[k*V + j];
        float t2 = m[i*V + j];
    
        m[i*V + j] = (t1 < t2) ? t1 : t2;
    }
    
    //calc(m,V,k,i,j);
}